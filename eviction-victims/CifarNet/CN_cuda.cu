#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////
//		Author:    Chethan Palangotu Keshava
//		LinkedIn:  https://www.linkedin.com/in/chethankeshava/
//		File:      CUDA implementation of CifarNet
//		Objective: Testing the performance of GPU architecture modifications done 
//			   to GPGPU-SIM. The simulator is built on old CUDA version (4.0)
//			   and hence no libraries are used for computations, with each
//			   computation done manually
/////////////////////////////////////////////////////////////////////////////////////////  
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <assert.h>
using namespace std;


extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);
void NeuralNetwork();

unsigned g_verbose;
unsigned NUM;

/////////////////////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	int i, commandline_error;
	commandline_error = 0;
	g_verbose = 0;
	if (argc >= 2) {
		NUM = atoi(argv[1]);
		for (i=2; i < argc;i++) {
			if (argv[i][0] == '-') {
				switch (argv[i][1]) {
				case 'v': g_verbose = 1;
					break;
				default: commandline_error=1;
				}
			}
			else commandline_error=1;
		}
	} else commandline_error=1;

	if (commandline_error || !NUM) {
		printf("Usage: ./NN <NUM> [-v]\n");
		printf("where NUM is the number of images to process in parallel (up to 10000 for the t10k-images-idx3-ubyte database file) and -v is used to display approximately what each image looks like.\n");
		return 1;
	}


	NeuralNetwork();
    //CUT_EXIT(argc, argv);
}


/////////////////////////////////////////////////////////////////////////////////////////
// Read all the weights from the weight files for all layers to the intialised host memory
/////////////////////////////////////////////////////////////////////////////////////////
void InitHostMem(double *Layer1_Weights_CPU, double *Layer2_Weights_CPU, double *Layer3_Weights_CPU, double *Layer4_Weights_CPU, double *Layer5_Weights_CPU)
{
	// initial layer 1 weight
	FILE * pFile1 = fopen ("data/conv1.txt","rb");
	if (pFile1 != NULL)
	{
		//printf("File Opened\n");
		char s[300000] = "";
		fread(s,sizeof(s),1,pFile1);
		//printf("Done2\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer1_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==2400)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile1);
	}	
	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 2 Weights
	FILE * pFile2 = fopen ("data/conv2.txt","rb");
	if (pFile2 != NULL)
	{
		//printf("File 2 Opened\n");
		char s[3000000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer2_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==25600)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile2);
	}	
	
	if (!pFile2)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 3 Weights
	FILE * pFile3 = fopen ("data/conv3.txt","rb");
	if (pFile3 != NULL)
	{
		//printf("File 3 Opened\n");
		char s[6000000] = "";
		fread(s,sizeof(s),1,pFile3);
		//printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer3_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==51200)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile3);
	}	
	
	if (!pFile3)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 4 Weights
	FILE * pFile4 = fopen ("data/ip1.txt","rb");
	if (pFile4 != NULL)
	{
		//printf("File 4 Opened\n");
		char s[8000000] = "";
		fread(s,sizeof(s),1,pFile4);
		//printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer4_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==65536)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile4);
	}	
	
	if (!pFile4)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Layer 5 Weights
	FILE * pFile5 = fopen ("data/ip2.txt","rb");
	if (pFile5 != NULL)
	{
		//printf("File 5 Opened\n");
		char s[80000] = "";
		fread(s,sizeof(s),1,pFile5);
		//printf("Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			Layer5_Weights_CPU[i] = temp_num;
			i++;
			index++;
			if(i==576)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile5);
	}	
	
	if (!pFile5)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Read the input image file, which is a txt file with R, G and B values
/////////////////////////////////////////////////////////////////////////////////////////
void LoadInput(int *Data_Layer_CPU)
{
	FILE * pFile1 = fopen ("data/speed-limit-35.txt","rb");
	if (pFile1 != NULL)
	{
		//printf("File Opened\n");
		char s[300000] = "";
		fread(s,sizeof(s),1,pFile1);
		//printf("Done2\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			int temp_num = atof(temp_string);	
			Data_Layer_CPU[i] = temp_num;
			i++;
			index++;
			if(i==(32*32*3))
			{
				//printf("Breaking input\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile1);
	}	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Split the RGB array to separate R, G amd B channel arrays
/////////////////////////////////////////////////////////////////////////////////////////
void ConvertInput(int *Data_Layer_CPU_R, int *Data_Layer_CPU_G, int *Data_Layer_CPU_B, int *Data_Layer_CPU)
{
	for(int i=0; i<32*32*3; i+=3)
	{
		Data_Layer_CPU_R[i/3] = Data_Layer_CPU[i];
		Data_Layer_CPU_G[i/3] = Data_Layer_CPU[i+1];
		Data_Layer_CPU_B[i/3] = Data_Layer_CPU[i+2];
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute first convolutional layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void ExecuteFirstLayer(double *Layer1_Weights_CPU, int *Data_Layer_CPU_R, int *Data_Layer_CPU_G, int *Data_Layer_CPU_B, double *Layer1_Features)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	for(int f=0; f<32; f++)
	{
				double result = 0;
				for(int i = x-2; i<=x+2; i++)
				{
    					for(int j=y-2; j<=y+2; j++)
    					{
						int x_index = i-x+2;
						int y_index = j-y+2;
						int m = (y_index)+(x_index)*5;
         					if(i<0 || j<0)
						{
							result+= 0;						
						}
         					else if(j>31 || i>31)
						{
							result+= 0;
						}
         					else
						{
							result += Data_Layer_CPU_R[(y_index-2) + x*32 + y + (x_index-2)*32]*Layer1_Weights_CPU[m+f*75] + Data_Layer_CPU_G[(y_index-2) + x*32 + y + (x_index-2)*32]*Layer1_Weights_CPU[m+25+f*75] + Data_Layer_CPU_B[(y_index-2) + x*32 + y + (x_index-2)*32]*Layer1_Weights_CPU[m+50+f*75];			
						}
					}
				} 
				Layer1_Features[f*32*32+x*32+y] = result;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute second convolutional layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void ExecuteSecondLayer(double *Layer2_Weights_CPU, double *Layer2_Features, double *Layer2_pool_GPU)
{
	double Features = 0;
	int x = threadIdx.x;
	int y = threadIdx.y;
	for(int f=0; f<32; f++)
	{
		Features = 0;
		for(int n=0; n<32; n++)
		{
			if(x<16)
			{
				if(y<16)
				{
					double result = 0;
					for(int i = x-2; i<=x+2; i++)
					{
    						for(int j=y-2; j<=y+2; j++)
    						{
							int x_index = i-x+2;
							int y_index = j-y+2;
							int m = (y_index)+(x_index)*5;
         						if(i<0 || j<0)
							{
								result+=0;
							}
         						else if(j>15 || i>15)
							{
								result+=0;	
							}
         						else
							{
               							result+= Layer2_pool_GPU[n*16*16 + (x_index+x-2)*16 + (y_index+y-2)]*Layer2_Weights_CPU[m+f*25*32+n*25];			
							}
						}
					} 
					Features += result;
				}
			}
		}
		//ReLU activation function computation
		if(Features<0)
			Features = 0;
		Layer2_Features[f*16*16 + x*16 + y] = Features;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute third convolutional layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void ExecuteThirdLayer(double *Layer3_Weights_CPU, double *Layer3_Features, double *Layer3_pool_GPU)
{
	double Features = 0;
	int x = threadIdx.x;
	int y = threadIdx.y;
	for(int f=0; f<64; f++)
	{
		Features = 0;
		for(int n=0; n<32; n++)
		{
			if(x<8)
			{
				if(y<8)
				{
					double result = 0;
					for(int i = x-2; i<=x+2; i++)
					{
    						for(int j=y-2; j<=y+2; j++)
    						{
							int x_index = i-x+2;
							int y_index = j-y+2;
							int m = (y_index)+(x_index)*5;
         						if(i<0 || j<0)
							{
								result+=0;
							}
         						else if(j>7 || i>7)
							{
								result+=0;
							}
         						else
							{
               							result+= Layer3_pool_GPU[n*8*8 + (x_index+x-2)*8 + (y_index+y-2)]*Layer3_Weights_CPU[m+f*25*32+n*25];			
							}
						}
					} 
					Features += result;
				}
			}
		}
		//ReLU activation function computation
		if(Features<0)
			Features = 0;
		Layer3_Features[f*8*8 + x*8 + y] = Features;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute fourth layer, which is a fully connected layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void ExecuteFourthLayer(double *Layer4_Weights_CPU, double *Layer4_Features, double *Pool3_Layer_Features)
{
	int n = threadIdx.x;
	{
		double result = 0;
		for(int f=0; f<64; f++)
		{
			for(int x=0; x<4; x++)
			{
				for(int y=0; y<4; y++)
				{
					result+= Pool3_Layer_Features[f*4*4 +x*4 + y] * Layer4_Weights_CPU[y+(x*4)+(f*4*4)+(n*4*4*64)];
				}
			}
		}
		Layer4_Features[n] = result;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute fifth layer, which is a fully connected layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void ExecuteFifthLayer(double *Layer5_Weights_CPU, double *Layer5_Features, double *Layer4_Features)
{
	int n = threadIdx.x;
	if(n<9)
	{
		double result = 0;
		for(int f=0; f<64; f++)
		{
			result+= Layer4_Features[f] * Layer5_Weights_CPU[f+n*64];
		}
		Layer5_Features[n] = result;
		result = 0;
	}
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute max pooling compuation for the first layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void pooling1(double *Layer2_Neurons_GPU,double *Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    int row = threadIdx.x;
    int col = threadIdx.y;
    double max = 0.0;
    {
        for(int output =0;output < out ;output++)
        {
            if(row%2 != 0)
            { 
                if(col%2 != 0)
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>31) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>31) break;
                            if(max < ((Layer2_Neurons_GPU[output*32*32+i*32+j])))
                                max =   ((Layer2_Neurons_GPU[output*32*32+i*32+j])) ;

                        }
                    }
		    //ReLU activation function compuation
		    if(max<0)
			max = 0;
                    Layer2_pool_GPU[output*16*16+(row-1)*8+(col-1)/2] = max;  
                    max = 0.0;   
                }
            }
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute average pooling compuation for the second layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void pooling2(double *Layer2_Neurons_GPU,double *Layer2_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    double avg = 0.0;
    int count = 0;
    int row = threadIdx.x;
    int col = threadIdx.y;
    {
        for(int output =0;output < out ;output++)
        {
            if((row%2 != 0) && (row<16))
            { 
                if((col%2 != 0) && (col<16))
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>15) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>15) break;
                            avg+= Layer2_Neurons_GPU[output*16*16 + i*16 + j];
			    count = count + 1;

                        }
                    }
                    Layer2_pool_GPU[output*8*8+(row-1)*4+(col-1)/2] = avg/count;    
                    avg = 0.0;   
		    count=0;
                }
            }
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////
// Device function to execute average pooling compuation for the third layer
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void pooling3(double *Layer3_Neurons_GPU,double *Layer3_pool_GPU,int out,int out_fr,int out_fc,int kernel,int stride_width,int in_fr,int in_fc)
{
    double avg = 0.0;
    int count = 0;
    int row = threadIdx.x;
    int col = threadIdx.y;
    {
        for(int output =0;output < out ;output++)
        {
            if((row%2 != 0) && (row<8))
            { 
                if((col%2 != 0) && (col<8))
                {
                    for(int i = row-1; i <= row+1; i++)
                    {   
			if(i>7) break;        
                        for(int j = col-1; j <= col+1; j++)
                        {
			    if(j>7) break;
                            avg+= ((Layer3_Neurons_GPU[output*8*8 + i*8 + j]));
			    count++;

                        }
                    }
                    Layer3_pool_GPU[output*4*4+(row-1)*2+(col-1)/2] = avg/count;   
                    avg = 0.0;   
		    count=0;
                }
            }
        }
    }
}

void NeuralNetwork()
{
	hipError_t err;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int deviceCount;                                                         
	hipGetDeviceCount(&deviceCount);                
	if (deviceCount == 0) {                                                  
		fprintf(stderr, "There is no device.\n");                            
		exit(EXIT_FAILURE);                                                  
	}                                                                        
	int dev;                                                                 
	for (dev = 0; dev < deviceCount; ++dev) {                                
		hipDeviceProp_t deviceProp;                                           
		hipGetDeviceProperties(&deviceProp, dev);   
		if (deviceProp.major >= 1)                                           
			break;                                                           
	}                                                                        
	if (dev == deviceCount) {                                                
		fprintf(stderr, "There is no device supporting CUDA.\n");            
		exit(EXIT_FAILURE);                                                  
	}                                                                        
	else                                                                     
		hipSetDevice(dev);
	//Allocation of host memory for weights
	double *Layer1_Weights_CPU = (double*) malloc (3*32*32* NUM * sizeof(double));
	double *Layer2_Weights_CPU = (double*) malloc (5*5*32*32* NUM * sizeof(double));
	double *Layer3_Weights_CPU = (double*) malloc (5*5*32*64* NUM * sizeof(double));
	double *Layer4_Weights_CPU = (double*) malloc (64*4*4*64* NUM * sizeof(double));
	double *Layer5_Weights_CPU = (double*) malloc (64*9* NUM * sizeof(double));

	printf("Program Start\n");
	//Allocation of host memory for input data
	int *Data_Layer_CPU_R = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU_G = (int*) malloc (32*32*NUM*sizeof(int));
	int *Data_Layer_CPU_B = (int*) malloc (32*32*NUM*sizeof(int));
	//Allocation of device memory for input data
	int *Data_Layer_GPU_R;
	int *Data_Layer_GPU_G;
	int *Data_Layer_GPU_B;
	int *Data_Layer_CPU = (int*) malloc (3*32*32*NUM*sizeof(int));
	InitHostMem(Layer1_Weights_CPU, Layer2_Weights_CPU, Layer3_Weights_CPU, Layer4_Weights_CPU, Layer5_Weights_CPU);
	LoadInput(Data_Layer_CPU);
	ConvertInput(Data_Layer_CPU_R, Data_Layer_CPU_G, Data_Layer_CPU_B, Data_Layer_CPU);
	double *Layer1_Features;
	double *Layer1_Weights_GPU;
	hipEventRecord(start);
	err = hipMalloc((void**) &Layer1_Features, 32*32*32* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Layer1_Weights_GPU, 2400* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Data_Layer_GPU_R, 32*32* NUM * sizeof(int));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Data_Layer_GPU_G, 32*32* NUM * sizeof(int));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMalloc((void**) &Data_Layer_GPU_B, 32*32* NUM * sizeof(int));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	printf("Malloc completed\n");
	hipMemcpy(Layer1_Weights_GPU,Layer1_Weights_CPU, sizeof(double)*2400*NUM, hipMemcpyHostToDevice);
	hipMemcpy(Data_Layer_GPU_R,Data_Layer_CPU_R, 32*32* NUM * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(Data_Layer_GPU_G,Data_Layer_CPU_G, 32*32* NUM * sizeof(int), hipMemcpyHostToDevice);
	err = hipMemcpy(Data_Layer_GPU_B,Data_Layer_CPU_B, 32*32* NUM * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	printf("Memcpy completed\n");
	dim3 n_threads(32,32,1);
	dim3 n_blocks(1,1,1); 
	hipDeviceSynchronize();
	//Execute First Layer
	ExecuteFirstLayer<<<n_blocks,n_threads>>>(Layer1_Weights_GPU, Data_Layer_GPU_R, Data_Layer_GPU_G, Data_Layer_GPU_B, Layer1_Features);
	
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "1st LayerKernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	hipDeviceSynchronize();
	double *Pool_Layer_Features;
	err = hipMalloc((void**) &Pool_Layer_Features, 32*16*16* NUM * sizeof(double));
	pooling1<<<n_blocks,n_threads>>>(Layer1_Features, Pool_Layer_Features, 32, 16, 16, 5, 2, 32, 32);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "1st Pool Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer2_Weights_GPU;
	err = hipMalloc((void**) &Layer2_Weights_GPU, 5*5*32*32* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer2_Weights_GPU,Layer2_Weights_CPU, sizeof(double)*5*5*32*32*NUM, hipMemcpyHostToDevice);
	double *Layer2_Features;	
	err = hipMalloc((void**) &Layer2_Features, 32*16*16* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	//Execute Second Layer
	ExecuteSecondLayer<<<n_blocks,n_threads>>>(Layer2_Weights_GPU, Layer2_Features, Pool_Layer_Features);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "2nd Layer Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Pool2_Layer_Features;
	hipMalloc((void**) &Pool2_Layer_Features, 32*8*8* NUM * sizeof(double));	
	pooling2<<<n_blocks,n_threads>>>(Layer2_Features, Pool2_Layer_Features, 32, 8, 8, 5, 2, 16, 16);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "2nd Pool Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer3_Weights_GPU;
	err = hipMalloc((void**) &Layer3_Weights_GPU, 5*5*32*64* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer3_Weights_GPU,Layer3_Weights_CPU, sizeof(double)*5*5*32*64*NUM, hipMemcpyHostToDevice);
	double *Layer3_Features;	
	err = hipMalloc((void**) &Layer3_Features, 64*8*8* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	//Execute Third Layer
	ExecuteThirdLayer<<<n_blocks,n_threads>>>(Layer3_Weights_GPU, Layer3_Features, Pool2_Layer_Features);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "3rd Layer Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Pool3_Layer_Features;
	hipMalloc((void**) &Pool3_Layer_Features, 64*4*4* NUM * sizeof(double));
	pooling3<<<n_blocks,n_threads>>>(Layer3_Features, Pool3_Layer_Features, 64, 4, 4, 5, 2, 8, 8);
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "3rd Pool Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer4_Features;
	hipMalloc((void**) &Layer4_Features, 64*sizeof(double));
	double *Layer4_Weights_GPU;
	err = hipMalloc((void**) &Layer4_Weights_GPU, 64*4*4*64* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer4_Weights_GPU,Layer4_Weights_CPU, sizeof(double)*64*4*4*64*NUM, hipMemcpyHostToDevice);
	//Execute Fourth Layer
	ExecuteFourthLayer<<<1,64>>>(Layer4_Weights_GPU, Layer4_Features, Pool3_Layer_Features);	
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "4th Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer5_Features;
	hipMalloc((void**) &Layer5_Features, 9*sizeof(double));
	double *Layer5_Weights_GPU;
	err = hipMalloc((void**) &Layer5_Weights_GPU, 64*9* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	hipMemcpy(Layer5_Weights_GPU,Layer5_Weights_CPU, sizeof(double)*64*9*NUM, hipMemcpyHostToDevice);
	//Execute Fifth Layer
	ExecuteFifthLayer<<<1,32>>>(Layer5_Weights_GPU, Layer5_Features, Layer4_Features);	
	hipDeviceSynchronize();
	err = hipGetLastError();
        if (err != hipSuccess)
        {
        	fprintf(stderr, "5th Kernel execution failed (error code %s)\n", hipGetErrorString(err));
       		exit(EXIT_FAILURE);
        }
	double *Layer5_output_CPU = (double*) malloc (9* NUM * sizeof(double));
	//Copy result back to host memory
	hipMemcpy(Layer5_output_CPU, Layer5_Features, 9* NUM * sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipStreamSynchronize(0);
	printf("Final values of 9 outout neurons without softmax:\n");
	for(int i=0; i<9; i++)
		printf("%.8f\n",Layer5_output_CPU[i]);
	printf("Device Execution time: %fms\n",milliseconds);
	//Tested on Acer Predator with Nvidia GTX 1060, this program ran for 28.133ms
    /*hipFree(Data_Layer_GPU_R);
    hipFree(Data_Layer_GPU_G);
    hipFree(Data_Layer_GPU_B);

    hipFree(Pool_Layer_Features);
    hipFree(Layer1_Weights_GPU);
    hipFree(Layer1_Features);
    hipFree(Layer2_Weights_GPU);
    hipFree(Pool2_Layer_Features);
    hipFree(Layer2_Features);
    hipFree(Layer3_Weights_GPU);
    hipFree(Pool3_Layer_Features);
    hipFree(Layer3_Features);
    hipFree(Layer4_Features);
    hipFree(Layer4_Weights_GPU);
    hipFree(Layer5_Features);
    hipFree(Layer5_Weights_GPU);*/

    free(Layer1_Weights_CPU);
    free(Layer2_Weights_CPU);
    free(Layer3_Weights_CPU);
    free(Layer4_Weights_CPU);
    free(Layer5_Weights_CPU);

    free(Data_Layer_CPU_R);
    free(Data_Layer_CPU_G);
    free(Data_Layer_CPU_B);
    free(Data_Layer_CPU);
}



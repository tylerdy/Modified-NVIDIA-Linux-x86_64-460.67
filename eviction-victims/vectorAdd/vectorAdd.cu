#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    //hipStream_t stream;
    //checkCudaErrors(hipStreamCreate(&stream));
    // for(int iter = 0; iter < 1024; iter++){
        // Print the vector length to be used, and compute its size
        int numElements = 50000;
        size_t size = numElements * sizeof(float);
        // printf("[Vector addition of %d elements]\n", numElements);

        // Allocate the host input vector A
        float *h_A = (float *)malloc(size);

        // Allocate the host input vector B
        float *h_B = (float *)malloc(size);

        // Allocate the host output vector C
        float *h_C = (float *)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }
        
        // Initialize the host input vectors
        for (int i = 0; i < numElements; ++i)
        {
            h_A[i] = rand()/(float)RAND_MAX;
            h_B[i] = rand()/(float)RAND_MAX;
        }

        float *start_addr = NULL;
        err = hipMalloc((void **)&start_addr, size*3);
        // printf("1\n");
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device start addr (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        
        // Allocate the device input vector A
        float *d_A = start_addr;
        // err = hipMalloc((void **)&d_A, size);
        // // printf("1\n");
        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // Allocate the device input vector B
        float *d_B = (float*)((void*)d_A + size);
        float *d_C = (float*)((void*)d_B + size);
        // err = hipMalloc((void **)&d_B, size);
        // // printf("2\n");
        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // // Allocate the device output vector C
        // float *d_C = NULL;
        // err = hipMalloc((void **)&d_C, size);
        // // printf("3\n");
        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
        // printf("Copy input data from the host memory to the CUDA device\n");
        err = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);


        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        //checkCudaErrors(hipStreamSynchronize(stream));
        err = hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        //checkCudaErrors(hipStreamSynchronize(stream));
        // Launch the Vector Add CUDA Kernel
        int threadsPerBlock = 256;
        int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
        // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
        // for(int j = 0; j < 1024; j++){
            vectorAdd<<<blocksPerGrid, threadsPerBlock,0>>>(d_A, d_B, d_C, numElements);
            //checkCudaErrors(hipStreamSynchronize(stream));
        // }
        // err = hipGetLastError();

        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        // printf("Copy output data from the CUDA device to the host memory\n");
        // err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // Verify that the result vector is correct
        // for (int i = 0; i < numElements; ++i)
        // {
        //     if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        //     {
        //         fprintf(stderr, "Result verification failed at element %d!\n", i);
        //         exit(EXIT_FAILURE);
        //     }
        // }

        // printf("Test PASSED\n");

        // Free device global memory
        // err = hipFree(d_A);

        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // err = hipFree(d_B);

        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // err = hipFree(d_C);

        // if (err != hipSuccess)
        // {
        //     fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        //     exit(EXIT_FAILURE);
        // }

        // Free host memory
        free(h_A);
        free(h_B);
        free(h_C);
    // }
    hipStreamSynchronize(0);
    printf("Done.\n");
    return 0;
}

